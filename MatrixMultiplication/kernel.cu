#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <omp.h> // OpenMP for CPU parallelism

#define TPB 16

__global__ void blockStripeKernel(int* A, int* B, int* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;

    if (row < N && col < N) {
        for (int k = 0; k < N; k++) {
            sum += A[row * N + k] * B[col * N + k];
        }
        C[row * N + col] = sum;
    }
}

__global__ void blockStripeKernelShared(int* A, int* B, int* C, int N) {
    __shared__ int tileA[TPB][TPB];
    __shared__ int tileB[TPB][TPB];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;

    for (int t = 0; t < (N + TPB - 1) / TPB; t++) {
        // Load tiles into shared memory
        if (row < N && t * TPB + threadIdx.x < N) {
            tileA[threadIdx.y][threadIdx.x] = A[row * N + t * TPB + threadIdx.x];
        }
        else {
            tileA[threadIdx.y][threadIdx.x] = 0;
        }

        if (col < N && t * TPB + threadIdx.y < N) {
            tileB[threadIdx.y][threadIdx.x] = B[(t * TPB + threadIdx.y) * N + col];
        }
        else {
            tileB[threadIdx.y][threadIdx.x] = 0;
        }

        __syncthreads();

        // Compute partial result
        for (int k = 0; k < TPB; k++) {
            sum += tileA[threadIdx.y][k] * tileB[k][threadIdx.x];
        }

        __syncthreads();
    }

    // Write final result
    if (row < N && col < N) {
        C[row * N + col] = sum;
    }
}

void cpuMatrixMultiply(int* A, int* B, int* C, int N, int startRow, int endRow, int n_threads) {
    omp_set_num_threads(n_threads);
    for (int i = startRow; i < endRow; ++i) {
    #pragma omp parallel for collapse(1)
        for (int j = 0; j < N; ++j) {
            int sum = 0;
            for (int k = 0; k < N; ++k) {
                sum += A[i * N + k] * B[j * N + k];
            }
            C[i * N + j] = sum;
        }
    }
}

void transposeMatrix(const int* input, int* output, int N) {
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            output[j * N + i] = input[i * N + j];
        }
    }
}

void initializeMatrix(int* mat, int N) {
    for (int i = 0; i < N * N; i++) {
        mat[i] = i % 10 + 1;
    }
}

void printMatrix(int* mat, int N) {
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%d ", mat[i * N + j]);
        }
        printf("\n");
    }
}

void printMatrix(int* mat, int N, int maxSize) {
    if (N > maxSize) {
        printf("Matrix is too large to display completely. Displaying top-left %dx%d submatrix:\n", maxSize, maxSize);
        for (int i = 0; i < maxSize; ++i) {
            for (int j = 0; j < maxSize; ++j) {
                printf("%d ", mat[i * N + j]);
            }
            printf("\n");
        }
    }
    else {
        printf("Full Matrix:\n");
        for (int i = 0; i < N; ++i) {
            for (int j = 0; j < N; ++j) {
                printf("%d ", mat[i * N + j]);
            }
            printf("\n");
        }
    }
}




int main() {
    int n_threads = 1;
    int N = 8192;
    int loadProportion = N + 1;

    //_sleep(500);

    // Define GPU and CPU work ranges
    int cpuStartRow = 0;
    int cpuEndRow = N / loadProportion; // CPU handles first portion
    int gpuStartRow = N / loadProportion;
    int gpuEndRow = N; // GPU handles the rest

    double cpuStart;
    double cpuEnd;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Allocate host memory
    int* h_A = (int*)malloc(N * N * sizeof(int));
    int* h_B = (int*)malloc(N * N * sizeof(int));
    int* h_C = (int*)malloc(N * N * sizeof(int));
    int* tempMat = (int*)malloc(N * N * sizeof(int));

    initializeMatrix(h_A, N);
    initializeMatrix(tempMat, N);

    transposeMatrix(tempMat, h_B, N);

    // Allocate device memory
    int* d_A, * d_B, * d_C;
    hipEventRecord(start);
    hipMalloc(&d_A, N * N * sizeof(int));
    hipMalloc(&d_B, N * N * sizeof(int));
    hipMalloc(&d_C, N * N * sizeof(int));

    hipMemcpy(d_A, h_A, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * N * sizeof(int), hipMemcpyHostToDevice);

    // Prepare GPU launch parameters
    dim3 threadsPerBlock(TPB, TPB);
    dim3 numBlocks((N + TPB - 1) / TPB, (gpuEndRow - gpuStartRow + TPB - 1) / TPB);

    // Prepare CUDA stream for parallel execution
    hipStream_t stream;
    hipStreamCreate(&stream);

    printf("-------------------------\n");
    double startAttempt = omp_get_wtime();

    // Start CPU computation in parallel
    //============================================================

    // GPU works on the second part of the matrix
    blockStripeKernelShared << <numBlocks, threadsPerBlock, 0, stream >> > (d_A, d_B, d_C, N);
    hipEventRecord(stop, stream);

    // CPU works on the first part of the matrix
    cpuStart = omp_get_wtime();
    cpuMatrixMultiply(h_A, h_B, h_C, N, cpuStartRow, cpuEndRow, n_threads);
    cpuEnd = omp_get_wtime();
    //============================================================
    hipMemcpy(h_C + gpuStartRow * N, d_C + gpuStartRow * N, (gpuEndRow - gpuStartRow) * N * sizeof(int), hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    hipDeviceSynchronize();

    double endAttempt = omp_get_wtime();

    float gpuTime = 0;
    hipEventElapsedTime(&gpuTime, start, stop);

    // Copy GPU result back


    // Print results
    printf("CPU Execution Time: %f ms\n", (cpuEnd - cpuStart) * 1000);
    printf("-------------------------\n");
    printf("GPU Execution Time: %f ms\n", gpuTime);
    printf("-------------------------\n");
    //printf("Total time: %f ms\n", (endAttempt - startAttempt) * 1000);

    // Optionally print the matrix
    printMatrix(h_C, N, 10);



    // Clean up
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipStreamDestroy(stream);

    return 0;
}
